#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <opencv2\opencv.hpp>
#include <iostream>
#include "time.h"

using namespace std;
using namespace cv;


//Sobel���ӱ�Ե���˺���
__global__ void sobelInCuda(unsigned char* dataIn, unsigned char* dataOut, int imgHeight, int imgWidth)
{
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int yIndex = threadIdx.y + blockIdx.y * blockDim.y;
    int index = yIndex * imgWidth + xIndex;
    float Gx = 0;
    float Gy = 0;

    if (xIndex > 0 && xIndex < imgWidth - 1 && yIndex > 0 && yIndex < imgHeight - 1)
    {
        Gx = dataIn[(yIndex - 1) * imgWidth + xIndex + 1] + 2 * dataIn[yIndex * imgWidth + xIndex + 1] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[yIndex * imgWidth + xIndex - 1] + dataIn[(yIndex + 1) * imgWidth + xIndex - 1]);
        Gy = dataIn[(yIndex - 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex - 1) * imgWidth + xIndex] + dataIn[(yIndex - 1) * imgWidth + xIndex + 1]
            - (dataIn[(yIndex + 1) * imgWidth + xIndex - 1] + 2 * dataIn[(yIndex + 1) * imgWidth + xIndex] + dataIn[(yIndex + 1) * imgWidth + xIndex + 1]);
        
        if (Gx < 0) Gx = 0;
        if (Gx > 255) Gx = 255;

        if (Gy < 0) Gy = 0;
        if (Gy > 255) Gy = 255;

        dataOut[index] = sqrt(Gx * Gx + Gy * Gy);
    }
}

//Sobel���ӱ�Ե���OpenMP����
void sobelInOpenMP(Mat srcImg, Mat dstImg, int imgHeight, int imgWidth)
{
    float Gx = 0;
    float Gy = 0;
    #pragma omp parallel for private(Gx, Gy)
    for (int i = 1; i < imgHeight - 1; i++)
    {
        uchar* dataUp = srcImg.ptr<uchar>(i - 1);
        uchar* data = srcImg.ptr<uchar>(i);
        uchar* dataDown = srcImg.ptr<uchar>(i + 1);
        uchar* out = dstImg.ptr<uchar>(i);
        for (int j = 1; j < imgWidth - 1; j++)
        {
            Gx = (dataUp[j + 1] + 2 * data[j + 1] + dataDown[j + 1]) - (dataUp[j - 1] + 2 * data[j - 1] + dataDown[j - 1]);
            Gy = (dataUp[j - 1] + 2 * dataUp[j] + dataUp[j + 1]) - (dataDown[j - 1] + 2 * dataDown[j] + dataDown[j + 1]);

            if (Gx < 0) Gx = 0;
            if (Gx > 255) Gx = 255;

            if (Gy < 0) Gy = 0;
            if (Gy > 255) Gy = 255;

            out[j] = sqrt(Gx * Gx + Gy * Gy);
        }
    }
}

//Sobel���ӱ�Ե���CPU����
void sobel(Mat srcImg, Mat dstImg, int imgHeight, int imgWidth)
{
    float Gx = 0;
    float Gy = 0;

    for (int i = 1; i < imgHeight - 1; i++)
    {
        uchar* dataUp = srcImg.ptr<uchar>(i - 1);
        uchar* data = srcImg.ptr<uchar>(i);
        uchar* dataDown = srcImg.ptr<uchar>(i + 1);
        uchar* out = dstImg.ptr<uchar>(i);
        for (int j = 1; j < imgWidth - 1; j++)
        {
            //�����ٶȿ�
            Gx = (dataUp[j + 1] + 2 * data[j + 1] + dataDown[j + 1]) - (dataUp[j - 1] + 2 * data[j - 1] + dataDown[j - 1]);
            Gy = (dataUp[j - 1] + 2 * dataUp[j] + dataUp[j + 1]) - (dataDown[j - 1] + 2 * dataDown[j] + dataDown[j + 1]);
            

            if (Gx < 0) Gx = 0;
            if (Gx > 255) Gx = 255;

            if (Gy < 0) Gy = 0;
            if (Gy > 255) Gy = 255;

            out[j] = sqrt(Gx * Gx + Gy * Gy);
        }
    }
}


int main()
{
    //��ӡGPU����
    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount(&count);

    for (int i = 0; i < count; i++)
    {
        hipGetDeviceProperties(&prop, i);
        cout << "the information for the device : " << i << endl;
        cout << "name:" << prop.name << endl;
        cout << "the memory information for the device : " << i << endl;
        cout << "total global memory:" << prop.totalGlobalMem << endl;
        cout << "total constant memory:" << prop.totalConstMem << endl;
        cout << "threads in warps:" << prop.warpSize << endl;
        cout << "max threads per block:" << prop.maxThreadsPerBlock << endl;
        cout << "max threads dims:" << prop.maxThreadsDim[0] << "  " << prop.maxThreadsDim[1] <<
            "  " << prop.maxThreadsDim[2] << endl;
        cout << "max grid dims:" << prop.maxGridSize[0] << "  " <<
            prop.maxGridSize[1] << "  " << prop.maxGridSize[2] << endl;
        cout << endl;
    }


    Mat grayImg = imread("./dog.jpg", 0);

    int imgHeight = grayImg.rows;
    int imgWidth = grayImg.cols;

    Mat gaussImg;
    //��˹�˲�
    GaussianBlur(grayImg, gaussImg, Size(3, 3), 0, 0, BORDER_DEFAULT);

    //Sobel����CPUʵ��
    Mat dst(imgHeight, imgWidth, CV_8UC1, Scalar(0));

    clock_t cpu_start = clock();
    sobel(gaussImg, dst, imgHeight, imgWidth);
    clock_t cpu_finish = clock();
    double cpu_duration = (double)(cpu_finish - cpu_start) / CLOCKS_PER_SEC;
    cout << "CPU run time = " << cpu_duration << "seconds" << endl;

    //Sobel����OpenMPʵ��
    clock_t openmp_start = clock();
    sobelInOpenMP(gaussImg, dst, imgHeight, imgWidth);
    clock_t openmp_finish = clock();
    double openmp_duration = (double)(openmp_finish - openmp_start) / CLOCKS_PER_SEC;
    cout << "OpenMP run time = " << openmp_duration << "seconds" << endl;


    //CUDAʵ�ֺ�Ĵ��ص�ͼ��
    Mat dstImg(imgHeight, imgWidth, CV_8UC1, Scalar(0));

    //����GPU�ڴ�
    unsigned char* d_in;
    unsigned char* d_out;

    hipMalloc((void**)&d_in, imgHeight * imgWidth * sizeof(unsigned char));
    hipMalloc((void**)&d_out, imgHeight * imgWidth * sizeof(unsigned char));

    //����˹�˲����ͼ���CPU����GPU
    hipMemcpy(d_in, gaussImg.data, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 blocksPerGrid((imgWidth + threadsPerBlock.x - 1) / threadsPerBlock.x, (imgHeight + threadsPerBlock.y - 1) / threadsPerBlock.y);


    double gpu_duration = 0;
    clock_t gpu_start, gpu_finish;
    int cnt = 25;
    for (int i = 0; i < 50; i++)
    {
        gpu_start = clock();
        sobelInCuda <<< blocksPerGrid, threadsPerBlock >>> (d_in, d_out, imgHeight, imgWidth);
        hipDeviceSynchronize(); // ���ʹ��CPU��ʱ��ʽ��һ��Ҫ��ͬ������
        gpu_finish = clock();
        if(i >= (50 - cnt))
            gpu_duration += (double)(gpu_finish - gpu_start) / CLOCKS_PER_SEC;
    }

    cout << "CUDA run time = " << gpu_duration / cnt << "seconds" << endl;


   /* for (int i = 0; i < 50; i++)
    {
        //ʹ��event����ʱ��
        float elapsedTime;
        hipEvent_t start, stop;
        hipEventCreate(&start);    //����Event
        hipEventCreate(&stop);
        hipEventRecord(start, 0);    //��¼��ǰʱ��

        //���ú˺���
        sobelInCuda <<< blocksPerGrid, threadsPerBlock >>> (d_in, d_out, imgHeight, imgWidth);

        hipEventRecord(stop, 0);    //��¼��ǰʱ��
        hipEventSynchronize(stop);

        hipEventElapsedTime(&elapsedTime, start, stop);
        cout << "CUDA Run time =" << elapsedTime / 1000 << "seconds" << endl;
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }*/


    //��ͼ�񴫻�GPU
    hipMemcpy(dstImg.data, d_out, imgHeight * imgWidth * sizeof(unsigned char), hipMemcpyDeviceToHost);

    //�ͷ�GPU�ڴ�
    hipFree(d_in);
    hipFree(d_out);


    namedWindow("��Եͼ��", WINDOW_FREERATIO);
    imshow("��Եͼ��", dstImg);

    waitKey(0);
    destroyAllWindows();


    return 0;
}